#include "hip/hip_runtime.h"
#include<hip/hip_runtime_api.h>
#include<handler_cuda_error.h>
#include<Common.h>
#include <hipblas.h>
#include<stdio.h>
#include"dfp_cu.h"

__device__ double newton_solver(LogCost * cost, double *tt,double *rho,
                      double *x){
                      
            double er = 1;
            double tol=1e-12;
            int iter=0;
            double a;
            double xn;
            double grad, hess;
            while (er>tol){
                
                grad = cost->grad(x,tt,rho);
                hess = cost->hess(x,tt,rho);
                a = grad/hess;
                xn= *x - a ;
                er = abs( a / *x);
                *x=xn;
                iter++;
            }
            return *x; 
                      
 }
__global__ void my_newton(double *res,
                           double *tt,
                           double *rho,int n){

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    double x0 = 1e-5;
    if (x>=n)
        return;
    LogCost cost;
    res[x] =  newton_solver( &cost, &tt[x], rho, &x0);

    return ;

}
__device__ double dfp_solver(Cost * cost, double *tt,double *rho,
                      double *x0,double *w_last,double *C){
    int maxk=100;

    double rho0=0.55;
    const double sigma=0.4;
    double epsilon=1e-10;
    int k=0;
    double Hk=1.0;
    double x= *x0;
    double gk = 0.0;
    double dk = 0.0;

    while(k < maxk){

        gk = cost->grad(x0,w_last,tt,rho,C);
//        printf("gk = %.4f\n",gk);
//        printf(" k = %d\n",k);

        if(  (gk*gk) < epsilon )
            break;
        dk = -Hk * gk;
        int m=0;
        int mk=0;
        while( m < 15){
            double p = pow(rho0,m);
            double temp = (*x0) + p * dk;
            double temp1 = cost->cost(&temp,w_last,tt,rho,C);
            double temp2 = cost->cost(x0,w_last,tt,rho,C) + sigma * p * gk * dk;
            if( temp1 < temp2){
                mk=m;break;
        }
            m++;
        }
        double p = pow(rho0,mk);
        x = *x0 + p * dk;

        double sk= x - *x0;
        double yk=cost->grad(&x,w_last,tt,rho,C) - gk;
        Hk = sk / (yk);
        k++;
        *x0 = x;

    }

    return x;
}
__global__ void dfp_choose(double *res,int indss_s,int indss_e,
                           int indsb_s,int indsb_e,
                           int indsm_s, int indsm_e,
                           double *tt,
                           double *rho,
                           double *lb,double *ub,
                           Account * account,int n){

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    double x0 = 0.0;

    if (x>=n)
        return;
    if ((x < indss_e) && ( x>= indss_s)  ){
        Ss_cost cost_ss;
        res[x] =  dfp_solver( &cost_ss, &tt[x], rho,
                           &x0, &account->w_last[x], &account->C);
        if(res[x] < lb[x])
            res[x] = lb[x];
        if(res[x] > ub[x])
            res[x] = ub[x];
        return ;
    }
    if ( (x < indsb_e) && (x >= indsb_s)){
        Sb_cost cost_sb;
        res[x] =  dfp_solver( &cost_sb, &tt[x], rho,
                           &x0, &account->w_last[x], &account->C);
        if(res[x] < lb[x])
            res[x] = lb[x];
        if(res[x] > ub[x])
            res[x] = ub[x];
        return ;
    }
    if ( (x < indsm_e) && (x >= indsm_s)){

        Sm_cost cost_sm;
        res[x] =  dfp_solver( &cost_sm, &tt[x], rho,
                           &x0, &account->w_last[x], &account->C);
        if(res[x] < lb[x])
            res[x] = lb[x];
        if(res[x] > ub[x])
            res[x] = ub[x];
        return;
    }
    return ;





}

__global__ void callogcost(double *res,double *x_cu,int n){
    int x = blockIdx.x * blockDim.x + threadIdx.x;



    if (x>=n)
        return;
    res[x] =  log(x_cu[x]);

}
__global__ void calcost(double *res,double *x_cu,int indss_s,int indss_e,
                           int indsb_s,int indsb_e,
                           int indsm_s, int indsm_e,
                           Account * account,int n){

    int x = blockIdx.x * blockDim.x + threadIdx.x;



    if (x>=n)
        return;
    res[x] = 0.0;
    if ((x < indss_e) && ( x>= indss_s)  ){

        double temp = pow(( account->C * ( x_cu[x] -  account->w_last[x] )+142.1)/1004.0,2.0);
        res[x] = 0.01477 * exp(-temp) * max(( x_cu[x] -  account->w_last[x] ),0.0);

        return ;
    }
    if ( (x < indsb_e) && (x >= indsb_s)){
        double temp = pow(( account->C * ( x_cu[x] -  account->w_last[x] )+1621.0)/1627.0,2.0);
        res[x] = 0.02079 * exp(-temp) * max(( x_cu[x] -  account->w_last[x] ),0.0);

        return ;
    }
    if ( (x < indsm_e) && (x >= indsm_s)){

        double temp = pow(( account->C * ( x_cu[x] -  account->w_last[x] )+198.9)/648.0, 2.0);
        res[x] = 0.02079 * exp(-temp) * max(( x_cu[x] -  account->w_last[x] ),0.0);

        return;
    }
    return ;





}
