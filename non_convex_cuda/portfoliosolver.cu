#include "hip/hip_runtime.h"
#include "portfoliosolver.h"
#include<hip/hip_runtime_api.h>
#include<handler_cuda_error.h>
#include<Common.h>
#include <hipblas.h>
#include<stdio.h>
#include"kenel_function.h"
#include"dfp_cu.h"
double objective_log(dim3 &block,hipblasHandle_t handle,
                     double * sigma_cu, double *norm_cu,double *x_cu,int n);

portfoliosolver::portfoliosolver(){

    this->record = SolverRecord();

}


void show_res(double *s,int n){
    double *res = (double *)malloc(n*sizeof(double));
    CHECK(hipMemcpy(res,s,n*sizeof(double),hipMemcpyDeviceToHost));
    for (int i = 0 ; i< n;++i){
        printf("%.6f\t",res[i]);
    }

    free(res);
}
void matrix_times(hipblasHandle_t handle,double *f,double *g,int mf,int nf, int kg,double *res){

    hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,mf,kg,nf,
                 &alpha1 ,f,mf,g,nf,&beta,res,mf);
}
void matrix_timesT(hipblasHandle_t handle,double *f,double *g,int mf,int nf, int kg,double *res){

    hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,mf,kg,nf,
                 &alpha1 ,f,mf,g,nf,&beta,res,mf);
}
int portfoliosolver::SolveModel(const Option & opts){



    hipblasHandle_t handle;
    hipblasCreate(&handle);

    //prepare for choose portfolio
    Ss_cost *ss_cost; CHECK(hipMalloc(&ss_cost, sizeof(Ss_cost)));
    Sb_cost *sb_cost; CHECK(hipMalloc(&sb_cost, sizeof(Sb_cost)));
    Sm_cost *sm_cost; CHECK(hipMalloc(&sm_cost, sizeof(Sm_cost)));
//    int ss_s = model->n_Ss[0];
//    int ss_e = model->n_Ss[model->n_Ss.size()-1];
//    int sb_s = model->n_Sb[0];
//    int sb_e = model->n_Sb[model->n_Sb.size()-1];
//    int sm_s = model->n_Sm[0];
//    int sm_e = model->n_Sm[model->n_Sm.size()-1];



    //
    int m = model->m;
    int n = model->n;int n_size = n * sizeof(double);

    // norm
    double eps_pri = 0.0;
    double eps_dual = 0.0;
    double r_norm = 0.0;
    double s_norm = 0.0;
    double sqrtn = sqrt(n)*opts.ABSTOL;
    double *norm_cu;CHECK(hipMalloc(&norm_cu,n_size ));
    double *norm_cu2;CHECK(hipMalloc(&norm_cu2,n_size ));

    double *z_cu,*z_old;
    double *u_cu;
    double *u_new;

    double *f_cu;
    double *R_cu;
    double *P_cu, *Pt_cu;
    double *Pb_cu, *x2_cu;

    double *APt_cu;
    double *PAt_cu;
    double *invs_cu;
    double *S_cu;
    double * Ainv;
    int * info ;


    double *tt_cu;
    CHECK(hipMalloc(&tt_cu,n*sizeof(double)));

    double *b_cu;
    CHECK(hipMalloc(&b_cu,n*sizeof(double)));

    CHECK(hipMalloc(&info, sizeof(int)));

    CHECK(hipMalloc(&z_cu,n*sizeof(double)));
    CHECK(hipMalloc(&z_old,n*sizeof(double)));
    double *x_cu, *x_hat_cu;
    CHECK(hipMalloc(&x_cu,n*sizeof(double)));
    CHECK(hipMalloc(&x_hat_cu,n*sizeof(double)));

    CHECK(hipMalloc(&Ainv,m*m*sizeof(double)));



    CHECK(hipMalloc(&u_cu,n*sizeof(double)));
    CHECK(hipMalloc(&u_new,n*sizeof(double)));
    CHECK(hipMalloc(&f_cu,n*sizeof(double)));
    CHECK(hipMalloc(&R_cu,n*sizeof(double)));

    CHECK(hipMalloc(&invs_cu,n*n*sizeof(double)));
    CHECK(hipMalloc(&Pb_cu,n*sizeof(double)));
    CHECK(hipMalloc(&x2_cu,n*sizeof(double)));

    CHECK(hipMalloc(&APt_cu,n*m*sizeof(double)));

    CHECK(hipMalloc(&PAt_cu,n*m*sizeof(double)));

    CHECK(hipMemcpy(R_cu,this->model->Er.data(),n*sizeof(double),hipMemcpyHostToDevice));


    Pt_cu = this->Pt_cu_;
    P_cu = this->P_cu_;
    S_cu = this->S;



    APt_cu = this->APt_cu_;
    PAt_cu = this->PAt_cu_;

    CHECK(hipMemcpy(b_cu,this->model->b.data(),n*sizeof(double),hipMemcpyHostToDevice));


    double *sigma_cu; CHECK(hipMalloc(&sigma_cu,n*n*sizeof(double)));
    CHECK(hipMemcpy(sigma_cu,this->model->Sigma.data(),n*n*sizeof(double),hipMemcpyHostToDevice));


    double *lb_cu;CHECK(hipMalloc(&lb_cu,n*sizeof(double)));
    double *ub_cu;CHECK(hipMalloc(&ub_cu,n*sizeof(double)));

    double *l0_cu;CHECK(hipMalloc(&l0_cu,n_size ));
    double *lhat0_cu;CHECK(hipMalloc(&lhat0_cu,n_size ));
    double *Bv0_cu;CHECK(hipMalloc(&Bv0_cu,n_size ));
    double *Au0_cu;CHECK(hipMalloc(&Au0_cu,n_size ));
    double *lhat_cu;CHECK(hipMalloc(&lhat_cu,n_size ));
    double *Bv_cu;CHECK(hipMalloc(&Bv_cu,n_size ));
    double *Au_cu;CHECK(hipMalloc(&Au_cu,n_size ));



    CHECK(hipMemcpy(lb_cu,this->model->lb.data(),n*sizeof(double),hipMemcpyHostToDevice));
    CHECK(hipMemcpy(ub_cu,this->model->ub.data(),n*sizeof(double),hipMemcpyHostToDevice));

    double *temp1_cu; // m * n
    double *temp2_cu; // m * m
    CHECK(hipMalloc(&temp1_cu,m * n*sizeof(double)));
    CHECK(hipMalloc(&temp2_cu, m * m*sizeof(double)));

    int freq = opts.adp_freq;
    int siter = std::max(opts.adp_start_iter-1, 1);
    int eiter = std::min(opts.adp_end_iter, opts.max_iter)+1;


    double *gamma_;CHECK(hipMalloc(&gamma_, sizeof(double)));
    hipMemcpy(gamma_, &opts.gamma,  sizeof(double), hipMemcpyHostToDevice);


    double *rho;
    double rhotemp = 1.0;
    CHECK(hipMalloc(&rho, sizeof(double)));
    hipMemcpy(rho, &rhotemp,  sizeof(double ), hipMemcpyHostToDevice);

    hipblasStatus_t stat;

    double **Ap, **Aip;
    double **A1 ,**Ai;
    CHECK(hipMalloc(&Ap, sizeof(double *)));
    A1 = (double **)malloc(sizeof(double *));
    CHECK(hipMalloc(&Aip, sizeof(double *)));
    Ai = (double **)malloc(sizeof(double *));
    A1[0] = temp2_cu;
    Ai[0] = Ainv;
    hipMemcpy(Ap, A1,  sizeof(double *), hipMemcpyHostToDevice);
    hipMemcpy(Aip, Ai,  sizeof(double *), hipMemcpyHostToDevice);


    Account *account_cu;CHECK(hipMalloc(&account_cu, sizeof(Account)));
    //CHECK(hipMemcpy(&account_cu->C, &this->model->C, sizeof(double),hipMemcpyHostToDevice));
    //CHECK(hipMemcpy(&account_cu->w_last[0], this->model->w_last, n*sizeof(double),hipMemcpyHostToDevice));

    dim3 block(n,n/128+ 1);

    printf("start solving");
    int iter = 0;
    for (iter = 1; iter < opts.max_iter;++iter){


        update_f<<<block,128>>>(f_cu,R_cu,z_cu,u_cu,rho,n);


        // Pb_b = - P_cu * f
        hipblasDgemv(handle, HIPBLAS_OP_N, n, n, &alpha_1,
                    P_cu, n, f_cu, 1, &beta, Pb_cu,1);

        // create invs n * n
        generate_invs<<<block,128>>>(invs_cu,S_cu, rho, n);

        // temp1 = APt*invs  m*n
        matrix_times(handle,APt_cu,invs_cu,m,n,n,temp1_cu);

        // temp2 = temp1*PAt
        matrix_times(handle,temp1_cu,PAt_cu,m,n,m,temp2_cu);

        // temp2 = inv(temp)
        stat = cublasDmatinvBatched(handle, m,  Ap, m,  Aip, m, info, 1);

        // temp_b = APt * invs * Pb_b = temp1_cu * Pb_b n*1
        hipMemcpy(x2_cu, b_cu,  n * sizeof(double), hipMemcpyDeviceToDevice);
        hipblasDgemv(handle, HIPBLAS_OP_N, m, n, &alpha1,
                    temp1_cu, m, Pb_cu, 1, &alpha_1, x2_cu,1);

        // x2 = Ainv * temp_b  m * 1
        hipblasDgemv(handle, HIPBLAS_OP_N, m, m, &alpha1,
                    Ainv, m, x2_cu, 1, &beta, x2_cu,1);

        // Pb_cu = Pb - PAt*x2
        hipblasDgemv(handle, HIPBLAS_OP_N, n, m, &alpha_1,
                    PAt_cu, n, x2_cu, 1, &alpha1, Pb_cu,1);
        hipblasDgemv(handle, HIPBLAS_OP_N, n, n, &alpha_1,
                    invs_cu, n, Pb_cu, 1, &beta, Pb_cu,1);
        hipblasDgemv(handle, HIPBLAS_OP_N, n, n, &alpha_1,
                    Pt_cu, n, Pb_cu, 1, &beta, x_cu,1);


        // zold = z
        hipMemcpy(z_old, z_cu,  n_size, hipMemcpyDeviceToDevice);

        // x_hat = gamma_*x + (1 - gamma_)*zold;
        update_x_hat<<<block,128>>>(x_hat_cu, x_cu,z_old,gamma_,n);

        // tt_cu =   x_hat_cu + u/rho;
        update_tt_cu<<<block,128>>>(tt_cu,x_hat_cu,u_cu,rho,n);

        // update z = min(max(x_hat + u/rho ,lb),ub) ;
        // for log problem there is no need
        
        //update_z_cu<<<block,128>>>(z_cu,tt_cu,lb_cu,ub_cu,n);


        //dfp_choose<<<block,128>>>(z_cu,ss_s,ss_e,sb_s,sb_e,sm_s,sm_e,tt_cu,rho,
        //                          lb_cu,ub_cu,account_cu,n);

        hipblasDnrm2(handle,n,z_cu,1,&r_norm); //r_norm
//        if( isnan(r_norm))  {
//            printf("z_cu  ");
//            show_res(z_cu,n);
//            printf("nan ! error");
//            exit(-1);
//        }

        my_newton<<<block,128>>>(z_cu,
                            tt_cu,
                            rho,n);
        // unew = u + rho* (x_hat - z);
        update_u_new<<<block,128>>>(u_new, u_cu,rho,x_hat_cu,z_cu,n);

        // objective function do not need

        // norms
        // x_cu - z_cu; z_cu - zold_cu; norm(x) norm(z) norm(u_new)

        if(iter == 1){
             update_l0_hat0_and_so_on<<<block,128>>>(l0_cu,lhat0_cu,Bv0_cu,Au0_cu,
                                                     u_new,u_cu,x_hat_cu,z_old,
                                                     z_cu,rho,n);
        }else{
            if( ((iter%freq) == 0) && (iter > siter)  && (iter < eiter) ){
                update_hat_and_so_on<<<block,128>>>(lhat_cu,Bv_cu,Au_cu,u_cu,x_hat_cu,z_old,z_cu,rho,n);
                aradmm_estimate(handle,rho,gamma_,Au_cu,Au0_cu,
                                lhat_cu,lhat0_cu,Bv_cu,Bv0_cu,u_new,l0_cu,n);

                update_l0_hat0_and_so_on<<<block,128>>>(l0_cu,lhat0_cu,Bv0_cu,Au0_cu,
                                                        u_new,u_cu,x_hat_cu,z_old,
                                                        z_cu,rho,n);
            }
        }
        // u = unew
        hipMemcpy(u_cu, u_new,  n_size ,hipMemcpyDeviceToDevice);

        // calculate norms


        hipblasDnrm2(handle,n,x_cu,1,&eps_dual); //xnorm
        hipblasDnrm2(handle,n,z_cu,1,&eps_pri);  // znorm
        eps_pri = sqrtn + opts.RELTOL * max(eps_dual,eps_pri);
        hipblasDnrm2(handle,n,u_new,1,&eps_dual);
        eps_dual  = sqrtn + opts.RELTOL *eps_dual;

        // norm(x-z)
        prepare_for_norm<<<block,128>>>(norm_cu,norm_cu2,x_cu,z_cu,rho,z_old,n);

        hipblasDnrm2(handle,n,norm_cu2,1,&r_norm); //r_norm
        hipblasDnrm2(handle,n,norm_cu,1,&s_norm); //s_ norm

        if( isnan(r_norm))  {
            printf("x_cu  ");
            show_res(x_cu,n);

            printf("z_cu  ");
            show_res(z_cu,n);

            printf("norm_cu2  ");
            show_res(norm_cu2,n);

            printf("nan ! error");
            exit(-1);
        }
        printf("%3d\t%10.4f\t%10.4f\t%10.4f\t%10.4f\t  \n", iter,
                     r_norm ,  eps_pri ,
                     s_norm ,  eps_dual );



#ifdef _RECORD_

        this->record.errors[iter] = r_norm;
        double value = 0.0;

//        calculate the objective value
        //hipblasDgemv(handle, HIPBLAS_OP_N, n, n, &alpha1,
//                    sigma_cu, n, x_cu, 1, &beta, norm_cu,1);
        //hipblasDdot (handle,  n,x_cu, 1,norm_cu, 1,&this->record.values[iter]);
        //hipblasDdot (handle,  n,x_cu, 1,R_cu, 1,&value);

        //this->record.values[iter] -= value;
        this->record.values[iter] = objective_log(block,handle, sigma_cu,norm_cu,z_cu,n);

//        calcost<<<block,128>>>(norm_cu, x_cu,ss_s,ss_e,sb_s,sb_e,sm_s,sm_e,
//                                    account_cu,n);
        hipblasDasum(handle, n, norm_cu ,1, &value);
        //this->record.values[iter] += value;

         printf("value = %.4f\n",this->record.values[iter]);

#endif
        if ( (r_norm  <  eps_pri)  && (s_norm  <  eps_dual ))
            break;





    }

    //calculate objective
    CHECK(hipMemcpy(&this->result[0], x_cu, n_size,hipMemcpyDeviceToHost));





    return iter;
}
double objective_log(dim3 &block,hipblasHandle_t handle, double * sigma_cu, double *norm_cu,double *x_cu,int n){
    double result = 0.0;
    double value = 0.0;
    hipblasDgemv(handle, HIPBLAS_OP_N, n, n, &alpha1,
                sigma_cu, n, x_cu, 1, &beta, norm_cu,1);
    hipblasDdot (handle,  n,x_cu, 1,norm_cu, 1,&result);
    callogcost<<<block,128>>>(norm_cu, x_cu,n);
    hipblasDasum(handle, n, norm_cu ,1, &value);
    result += value;
    return result;
}
