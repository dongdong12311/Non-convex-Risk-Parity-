#include "hip/hip_runtime.h"
#include<hip/hip_runtime_api.h>
#include<handler_cuda_error.h>
#include<Common.h>
#include <hipblas.h>
#include<stdio.h>
#include"kenel_function.h"
#include<math.h>
const double gamma0 = 1.5;
const double gmh = 1.9 ;
const double gmg = 1.1 ;
const double minval = 1e-16;
const double orthval = 0.2;

double curv_adaptive_BB(double al_h,double de_h){

    double tmph = de_h/al_h;
    double tau_h;
    if (tmph > .8)
        tau_h = de_h;
    else
        tau_h = al_h - 0.8*de_h;



    return tau_h;

}
void aradmm_estimate(hipblasHandle_t handle,
                     double* rho,
                               double *gamma_,
                               double *Au,
                               double *Au0,
                               double *l_hat,
                               double *l_hat0,
                               double *Bv,
                               double *Bv0,
                               double *l,
                               double * l0,int n){

    //Au0 = -Au + Au0
    hipblasDaxpy(handle,n,&alpha_1,Au,1,Au0,1);
    hipblasDaxpy(handle,n,&alpha_1,Bv,1,Bv0,1);
    //lhat0 = - lhat + lhat0
    hipblasDaxpy(handle,n,&alpha_1,l_hat,1,l_hat0,1);
    hipblasDaxpy(handle,n,&alpha_1,l ,1,l0,1);

    double ul_hat = 0.0;
    hipblasDdot (  handle,   n,Au0, 1,
                               l_hat0, 1,&ul_hat);

    double vl = 0.0;
    hipblasDdot (  handle,   n,Bv0, 1,
                               l0, 1,&vl);
    //hipblasDasum(handle, n, Au0 ,1, &ul_hat);
    double dl = 0.0;
    double dl_hat = 0.0;
    hipblasDnrm2(  handle,   n, l_hat0, 1, &dl_hat);
    hipblasDnrm2(  handle,   n, l0, 1, &dl);

    double du = 0.0;
    double dv = 0.0;
    hipblasDnrm2(  handle,   n, Au0, 1, &du);
    hipblasDnrm2(  handle,   n, Bv0, 1, &dv);


//    printf("ul_hat=%.4f\n",ul_hat);
//    printf("vl=%.4f\n",vl);
//    printf("dl=%.4f\n",dl);
//    printf("dlhat=%.4f\n",dl_hat);
//    printf("du=%.4f\n",du);
//    printf("dv=%.4f\n",dv);
    bool hflag = false;
    bool gflag = false;
    double tau = 0.0;
    double bb_h,bb_g;
    if (ul_hat > orthval*du*dl_hat + minval){
        hflag = true;
        double al_h = dl_hat*dl_hat/ul_hat;
        double de_h = ul_hat/(du*du);
        bb_h = curv_adaptive_BB(al_h, de_h);
    }
    if (vl > orthval*dv*dl + minval){
        gflag = true;
        double al_g = dl*dl/vl;
        double de_g = vl/(dv*dv);
        bb_g = curv_adaptive_BB(al_g, de_g);
    }
    double gamma;
    if (hflag && gflag){
        double ss_h = sqrt(bb_h);
        double ss_g = sqrt(bb_g);
        gamma = std::min(1 + 2.0/(ss_h/ss_g + ss_g/ss_h), gamma0);
        tau = ss_h*ss_g;
    }else if (hflag){
        gamma = gmh;
        tau = bb_h;
    }
    else if (gflag){
        gamma = gmg;
        tau = bb_g;
    }
    else
        gamma = gamma0;

    if(hflag || gflag){
        hipMemcpy(rho,&tau,sizeof(double),hipMemcpyHostToDevice);
        hipMemcpy(rho,&tau,sizeof(double),hipMemcpyHostToDevice);
    }


}


__global__ void update_f(double*f,double*R,double * z,double *u,double *rho,int n){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x>=n)
        return;
    f[x] = -R[x] - *rho * z[x] + u[x];
}

__global__ void update_u_new(double *u_new, double * u_cu,double *rho,double *x_hat_cu,
                             double *z_cu,int n){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x>=n)
        return;
    u_new[x] =u_cu[x] + (*rho) *(x_hat_cu[x] - z_cu[x]);


}
__global__ void update_x_hat(double * xhat, double*x,double *zold,double *gamma,int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i>=n)
        return;
    xhat[i] = *gamma * x[i] + (1 - *gamma) * zold[i];
}
__global__ void update_tt_cu(double * tt, double *x_hat, double *u,double *rho,int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i>=n)
        return;
    tt[i] =  x_hat[i] +  u[i]/ (*rho);
}
__global__ void update_z_cu(double * z, double *tt, double * lb, double *ub,
                            int n){
    // z =   min(max(tt ,lb),ub) ;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i>=n)
        return;
    z[i] =  fmax(tt[i],lb[i]);
    z[i] =  fmin(z[i],ub[i]);

}
__global__ void generate_invs(double *f,double *d,double *rho,int n){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x>=n)
        return;
    f[x*n+x] = 1.0/(d[x] + *rho - 1.0);
}
__global__ void update_l0_hat0_and_so_on(double * l0_cu, double *lhat0_cu,
                         double *Bv0_cu,double *Au0_cu,
                                        double *u_new,double *u_cu,
                         double *x_hat_cu,double *z_old,
                                         double * z_cu,double *rho,int n){

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x>=n)
        return;
    l0_cu[x] = u_new[x];
    lhat0_cu[x] = u_cu[x] + (*rho) * (x_hat_cu[x] - z_old[x]);
    Bv0_cu[x] =  z_cu[x];
    Au0_cu[x] =  -x_hat_cu[x];
}

__global__ void update_hat_and_so_on(double *lhat0_cu,
                         double *Bv0_cu,double *Au0_cu,
                                         double *u_cu,
                         double *x_hat_cu,double *z_old,
                                         double * z_cu,double *rho,int n){

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x>=n)
        return;

    lhat0_cu[x] = u_cu[x] + (*rho) * (x_hat_cu[x] - z_old[x]);
    Bv0_cu[x] =  z_cu[x];
    Au0_cu[x] =  -x_hat_cu[x];
}
__global__ void prepare_for_norm(double *norm_cu,double *norm_cu2,double *x_cu,
                                 double *z_cu,double *rho,
                                 double *z_old,int n){


    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x>=n)
        return;
    norm_cu2[x]  = x_cu[x] - z_cu[x];
    norm_cu[x]  = *rho *(z_old[x] - z_cu[x]);

}
